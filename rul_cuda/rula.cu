#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

#include <cstring>
#include <algorithm>
#include <string>
#include <iostream>

using std::cout;
using std::endl;



void printDeviceInfo() { 
	hipDeviceProp_t deviceProp;

	int devID = 0;

	auto error = hipGetDeviceProperties(&deviceProp, devID);

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name,
		deviceProp.major, deviceProp.minor);
}


hipError_t checkCudaError(hipError_t cudaStatus, const char* message) {

	if (cudaStatus != hipError_t::hipSuccess) {
		printf( "Cuda error code: %d, '%s' msg: %s!\n", cudaStatus, message, hipGetErrorString(cudaStatus));
		throw cudaStatus;
	}
	return cudaStatus;
}

hipError_t checkCudaError(hipError_t cudaStatus) {
	return checkCudaError(cudaStatus, "");
}


hipError_t go(numeric_t** resultPtr, numeric_t** a, size_t size, int groups);



const char* concatStrs(const std::string desc, const std::string errStr) {
	return (desc + errStr).c_str();
}

const size_t CHUNK = 1000;
const numeric_t _min = -100;
const numeric_t _max = 100;
const numeric_t _delta = .000001;


int main(int argc, char** argv)
{

	printDeviceInfo();

	const size_t arraySize = ceil((_max - _min) / _delta);
	const size_t groupCount = arraySize % CHUNK == 0 ? arraySize / CHUNK : arraySize / CHUNK + 1;

	cout << "N = " << arraySize << "(" << arraySize / 1024.0 * sizeof(numeric_t) / 1000 / 1000 << " GB)" << endl;
	cout << "Allocating " << groupCount << " chunks of " << CHUNK * sizeof(numeric_t) << " bytes each " << endl;

	numeric_t** a = new numeric_t*[groupCount];
	numeric_t** result = new numeric_t*[groupCount];
	for (size_t i = 0; i < groupCount; i++) {
		a[i] = new numeric_t[CHUNK];
		result[i] = new numeric_t[CHUNK];
	}

	for (size_t i = 0; i < arraySize; i++) {
		size_t group = i / CHUNK;
		size_t x = i % CHUNK;
		a[group][x] = _delta * i + _min;
	}
	hipError_t cudaStatus;

	go(result, a, arraySize, groupCount);

	cout << endl << endl;
	cout << "INPUT || OUTPUT" << endl;
	//for (int r = 0; r < arraySize; r++) {
		//if (r%10==0)
		//cout << r << ". f(" << a[0][r] << ") = " << exp(a[0][r]) << " | " << result[0][r] << endl;
	//}
	cout << endl << endl;
	float rows = 100;
	for (int r = 0; r < rows; r++) {
		int i = r / rows * arraySize;
		size_t group = i / CHUNK;
		size_t x = i % CHUNK;
		cout << r << ". f(" << a[group][x] << ") = " << exp(a[group][x]) << " | " << result[group][x] << endl;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();

	for (int i = 0; i < groupCount; i++) {
		delete[] a[i];
		delete[] result[i];
	} 
	delete[] a;
	delete[] result;

	getchar();

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t go(numeric_t** resultPtr, numeric_t** a, size_t size, int groups)
{
	if (size == 0) {
		return hipError_t::hipSuccess;
	}
	numeric_t* input = 0;
	numeric_t* output = 0;
	numeric_t* blockSum = 0;
	hipError_t status = hipError_t::hipSuccess;

	try {
		hipSetDevice(0);

		cout << "Allocating memory on device. ";
		checkCudaError(hipMalloc((void**)&output, size * sizeof(numeric_t)), "hipMalloc output");
		checkCudaError(hipMalloc((void**)&input, size * sizeof(numeric_t)), "hipMalloc input");


		cout << "Copying memory to device. ";
		for (int i = 0; i < groups; i++) {
			numeric_t* addr = input + i * CHUNK;
			checkCudaError(hipMemcpy(addr, a[i], CHUNK * sizeof(numeric_t), hipMemcpyHostToDevice), "copying memory");
		}
		//hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		int B = std::max((size_t)1, (size_t)ceil((float)size / THREADS_IN_BLOCK));
		int SH = 0;// THREADS_IN_BLOCK * sizeof(numeric_t);

		checkCudaError(hipMalloc((void**)&blockSum, B), "hipMalloc cache");
		cout << "Blocks: " << B << " Threads: " << THREADS_IN_BLOCK << " Shared: " << SH << endl;

		// Launch a kernel on the GPU with one thread for each element.
		
		cout << "Calculating paramater values. " << endl;
		expKernel <<<B, THREADS_IN_BLOCK >>> (input, size);
		checkCudaError(hipGetLastError(), "expKernel launch failed: ");
		cout << "Calculating inf areas. " << endl;
		intKernel <<<B, THREADS_IN_BLOCK >>> (input, _delta, size); 
		checkCudaError(hipGetLastError(), "intKernel launch failed: ");
		cout << "Calculating partial sums. " << endl;
		sumKernel <<<B, THREADS_IN_BLOCK >>> (output, input, blockSum, size);
		checkCudaError(hipGetLastError(), "sumKernel launch failed: ");
		

		B = std::max((size_t)1, (size_t)ceil(B / THREADS_IN_BLOCK));

		cout << "Merging results. " << endl;
		sumBlocksKernel <<<B, THREADS_IN_BLOCK >>> (output, blockSum, size);
		checkCudaError(hipGetLastError(), "sumBlocksKernel launch failed: ");

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.

		hipDeviceSynchronize();
		// Copy output vector from GPU buffer to host memory.

		cout << "Retrieving results from device. " << endl;
		for (int i = 0; i < groups; i++) {
			hipMemcpy(resultPtr[i], output + i * CHUNK, CHUNK * sizeof(numeric_t), hipMemcpyDeviceToHost);
		}

	}
	catch (hipError_t e) {
		printf("hipGetLastError: %s\n", hipGetErrorString(e));
		status = e;
	}
	hipFree(input);
	hipFree(output);
	hipFree(blockSum);

	return status;
}
